#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include "histogram_eq_kernels.h"

namespace py = pybind11;

#define HISTOGRAM_SIZE 256
#define BLOCK_SIZE 256

// Wrapper function to perform histogram equalization
void histogramEqualizationWrapper(py::array_t<unsigned char> inputImage, py::array_t<unsigned char> outputImage, int width, int height) {
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // Access input and output arrays
    auto inputPtr = inputImage.mutable_data();
    auto outputPtr = outputImage.mutable_data();

    const int imageSize = width * height;

    // Allocate memory for the image on the host
    unsigned char *h_inputImage = inputPtr;
    unsigned char *h_outputImage = outputPtr;

    // Allocate device memory
    unsigned char *d_inputImage, *d_outputImage;
    float *d_cdf;

    hipMalloc((void **)&d_inputImage, imageSize * sizeof(unsigned char));
    hipMalloc((void **)&d_outputImage, imageSize * sizeof(unsigned char));
    hipMalloc((void **)&d_cdf, HISTOGRAM_SIZE * sizeof(float));

    // Copy the input image to the device
    hipMemcpy(d_inputImage, h_inputImage, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    int *d_histogram;
    hipMalloc((void **)&d_histogram, HISTOGRAM_SIZE * sizeof(int));

    // Set grid and block dimensions for histogram calculation
    dim3 histogramBlockDim(BLOCK_SIZE);
    dim3 histogramGridDim((imageSize + histogramBlockDim.x - 1) / histogramBlockDim.x);

    // Launch histogram calculation kernel
    calculateHistogram<<<histogramGridDim, histogramBlockDim>>>(d_inputImage, imageSize, d_histogram);

    // Set grid and block dimensions for CDF calculation
    dim3 cdfBlockDim(1);
    dim3 cdfGridDim(1);

    // Launch CDF calculation kernel
    calculateCDF_serial<<<cdfGridDim, cdfBlockDim>>>(d_histogram, imageSize, d_cdf);

    // Copy the CDF back to host
    // float cdf[HISTOGRAM_SIZE];
    // hipMemcpy(cdf, d_cdf, HISTOGRAM_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Set grid and block dimensions
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((imageSize + blockDim.x - 1) / blockDim.x);

    // Launch histogramEqualization kernel
    histogramEqualization<<<gridDim, blockDim>>>(d_inputImage, d_outputImage, imageSize, d_cdf);

    // Copy the result back to host
    hipMemcpy(h_outputImage, d_outputImage, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float totalTime;
    hipEventElapsedTime(&totalTime, start, stop);
    printf("Total Cpp runtime calculation time: %f ms\n", totalTime);

    // Free allocated memory on the device
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    hipFree(d_cdf);
    hipFree(d_histogram);
}

// Bindings for Python
PYBIND11_MODULE(cuda_hist_eq, m) {
    m.doc() = "CUDA Histogram Equalization";

    m.def("histogram_equalization", &histogramEqualizationWrapper, "Perform histogram equalization using CUDA");
}

